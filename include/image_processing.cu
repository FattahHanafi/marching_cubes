#include "hip/hip_runtime.h"
#include <cuda_device_runtime_api.h>
#include <thrust/detail/copy.h>
#include <thrust/detail/minmax.h>
#include <thrust/detail/raw_pointer_cast.h>
#include <thrust/extrema.h>
#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>

#include <cstdint>
#include <cstring>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <string>

#include "image_processing.hpp"

__global__ void d_initialize_A(float* d_A) {
  d_A[0] = 1.0 / 6.0;
  d_A[1] = 4.0 / 6.0;
  d_A[2] = 1.0 / 6.0;
  d_A[3] = 0.0 / 6.0;
  d_A[4] = -3.0 / 6.0;
  d_A[5] = 0.0 / 6.0;
  d_A[6] = 3.0 / 6.0;
  d_A[7] = 0.0 / 6.0;
  d_A[8] = 3.0 / 6.0;
  d_A[9] = -6.0 / 6.0;
  d_A[10] = 3.0 / 6.0;
  d_A[11] = 0.0 / 6.0;
  d_A[12] = -1.0 / 6.0;
  d_A[13] = 3.0 / 6.0;
  d_A[14] = -3.0 / 6.0;
  d_A[15] = 1.0 / 6.0;
}

__global__ void d_clear(float* d_GD, uint32_t* d_nums) {
  const uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  d_GD[idx] = 0.0f;
  d_nums[idx] = 0;
}

__global__ void d_initialize_U(float* d_U, const uint32_t N) {
  const uint32_t row = blockIdx.x;
  const uint32_t power = threadIdx.x;
  const uint32_t P = blockDim.x;

  d_U[row * P + power] = pow(float(row) / gridDim.x, power);
}

__global__ void d_evaluate_pixels(float* d_pixels, const float* d_U,
                                  const float* d_A, const float* d_P,
                                  const uint32_t P, const uint32_t N) {
  const uint32_t pixel_y = blockIdx.x;
  const uint32_t pixel_x = threadIdx.x;
  const uint32_t W = blockDim.x;

  uint32_t u_idx = pixel_x % (W / N) * (P + 1);
  uint32_t p_idx = pixel_y * (N * P + 1) + (pixel_x / (W / N)) * P;

  float val = 0.0;
  for (uint32_t i = 0; i < P + 1; ++i) {
    for (uint32_t j = 0; j < P + 1; ++j) {
      val += d_U[u_idx + i] * d_A[i * (P + 1) + j] * d_P[p_idx + j];
    }
  }
  d_pixels[pixel_y * W + pixel_x] = val;
}

__global__ void d_update_is_valid(bool* d_is_valid, const float* d_raw_pixels,
                                  const float th) {
  const uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  d_is_valid[idx] = d_raw_pixels[idx] > th;
}

__global__ void d_evaluate_error(const bool* d_is_valid,
                                 const float* d_raw_pixels,
                                 const float* d_pixels, float* d_error) {
  const uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  d_error[idx] =
      d_is_valid[idx] ? pow(d_pixels[idx] - d_raw_pixels[idx], 2.0f) : 0.0f;
}

__global__ void d_initialize_J(float* d_J, const float* d_U, const float* d_A,
                               const uint32_t P) {
  const uint32_t u_idx = threadIdx.x * (P + 1);
  const uint32_t j_idx = u_idx * (P + 2);

  d_J[j_idx + 0] = pow(d_A[0] * d_U[u_idx + 0] + d_A[4] * d_U[u_idx + 1] +
                           d_A[8] * d_U[u_idx + 2] + d_A[12] * d_U[u_idx + 3],
                       2.0) *
                   2.0;
  d_J[j_idx + 1] = (d_A[0] * d_U[u_idx + 0] + d_A[4] * d_U[u_idx + 1] +
                    d_A[8] * d_U[u_idx + 2] + d_A[12] * d_U[u_idx + 3]) *
                   (d_A[1] * d_U[u_idx + 0] + d_A[5] * d_U[u_idx + 1] +
                    d_A[9] * d_U[u_idx + 2] + d_A[13] * d_U[u_idx + 3]) *
                   2.0;
  d_J[j_idx + 2] = (d_A[0] * d_U[u_idx + 0] + d_A[4] * d_U[u_idx + 1] +
                    d_A[8] * d_U[u_idx + 2] + d_A[12] * d_U[u_idx + 3]) *
                   (d_A[2] * d_U[u_idx + 0] + d_A[6] * d_U[u_idx + 1] +
                    d_A[10] * d_U[u_idx + 2] + d_A[14] * d_U[u_idx + 3]) *
                   2.0;
  d_J[j_idx + 3] = (d_A[0] * d_U[u_idx + 0] + d_A[4] * d_U[u_idx + 1] +
                    d_A[8] * d_U[u_idx + 2] + d_A[12] * d_U[u_idx + 3]) *
                   (d_A[3] * d_U[u_idx + 0] + d_A[7] * d_U[u_idx + 1] +
                    d_A[11] * d_U[u_idx + 2] + d_A[15] * d_U[u_idx + 3]) *
                   2.0;
  d_J[j_idx + 4] =
      -2.0 * (d_A[0] * d_U[u_idx + 0] + d_A[4] * d_U[u_idx + 1] +
              d_A[8] * d_U[u_idx + 2] + d_A[12] * d_U[u_idx + 3]);  // * y
  d_J[j_idx + 5] = (d_A[0] * d_U[u_idx + 0] + d_A[4] * d_U[u_idx + 1] +
                    d_A[8] * d_U[u_idx + 2] + d_A[12] * d_U[u_idx + 3]) *
                   (d_A[1] * d_U[u_idx + 0] + d_A[5] * d_U[u_idx + 1] +
                    d_A[9] * d_U[u_idx + 2] + d_A[13] * d_U[u_idx + 3]) *
                   2.0;
  d_J[j_idx + 6] = pow(d_A[1] * d_U[u_idx + 0] + d_A[5] * d_U[u_idx + 1] +
                           d_A[9] * d_U[u_idx + 2] + d_A[13] * d_U[u_idx + 3],
                       2.0) *
                   2.0;
  d_J[j_idx + 7] = (d_A[1] * d_U[u_idx + 0] + d_A[5] * d_U[u_idx + 1] +
                    d_A[9] * d_U[u_idx + 2] + d_A[13] * d_U[u_idx + 3]) *
                   (d_A[2] * d_U[u_idx + 0] + d_A[6] * d_U[u_idx + 1] +
                    d_A[10] * d_U[u_idx + 2] + d_A[14] * d_U[u_idx + 3]) *
                   2.0;
  d_J[j_idx + 8] = (d_A[1] * d_U[u_idx + 0] + d_A[5] * d_U[u_idx + 1] +
                    d_A[9] * d_U[u_idx + 2] + d_A[13] * d_U[u_idx + 3]) *
                   (d_A[3] * d_U[u_idx + 0] + d_A[7] * d_U[u_idx + 1] +
                    d_A[11] * d_U[u_idx + 2] + d_A[15] * d_U[u_idx + 3]) *
                   2.0;
  d_J[j_idx + 9] =
      -2.0 * (d_A[1] * d_U[u_idx + 0] + d_A[5] * d_U[u_idx + 1] +
              d_A[9] * d_U[u_idx + 2] + d_A[13] * d_U[u_idx + 3]);  // * y
  d_J[j_idx + 10] = (d_A[0] * d_U[u_idx + 0] + d_A[4] * d_U[u_idx + 1] +
                     d_A[8] * d_U[u_idx + 2] + d_A[12] * d_U[u_idx + 3]) *
                    (d_A[2] * d_U[u_idx + 0] + d_A[6] * d_U[u_idx + 1] +
                     d_A[10] * d_U[u_idx + 2] + d_A[14] * d_U[u_idx + 3]) *
                    2.0;
  d_J[j_idx + 11] = (d_A[1] * d_U[u_idx + 0] + d_A[5] * d_U[u_idx + 1] +
                     d_A[9] * d_U[u_idx + 2] + d_A[13] * d_U[u_idx + 3]) *
                    (d_A[2] * d_U[u_idx + 0] + d_A[6] * d_U[u_idx + 1] +
                     d_A[10] * d_U[u_idx + 2] + d_A[14] * d_U[u_idx + 3]) *
                    2.0;
  d_J[j_idx + 12] = pow(d_A[2] * d_U[u_idx + 0] + d_A[6] * d_U[u_idx + 1] +
                            d_A[10] * d_U[u_idx + 2] + d_A[14] * d_U[u_idx + 3],
                        2.0) *
                    2.0;
  d_J[j_idx + 13] = (d_A[2] * d_U[u_idx + 0] + d_A[6] * d_U[u_idx + 1] +
                     d_A[10] * d_U[u_idx + 2] + d_A[14] * d_U[u_idx + 3]) *
                    (d_A[3] * d_U[u_idx + 0] + d_A[7] * d_U[u_idx + 1] +
                     d_A[11] * d_U[u_idx + 2] + d_A[15] * d_U[u_idx + 3]) *
                    2.0;
  d_J[j_idx + 14] =
      -2.0 * (d_A[2] * d_U[u_idx + 0] + d_A[6] * d_U[u_idx + 1] +
              d_A[10] * d_U[u_idx + 2] + d_A[14] * d_U[u_idx + 3]);  // * y
  d_J[j_idx + 15] = (d_A[0] * d_U[u_idx + 0] + d_A[4] * d_U[u_idx + 1] +
                     d_A[8] * d_U[u_idx + 2] + d_A[12] * d_U[u_idx + 3]) *
                    (d_A[3] * d_U[u_idx + 0] + d_A[7] * d_U[u_idx + 1] +
                     d_A[11] * d_U[u_idx + 2] + d_A[15] * d_U[u_idx + 3]) *
                    2.0;
  d_J[j_idx + 16] = (d_A[1] * d_U[u_idx + 0] + d_A[5] * d_U[u_idx + 1] +
                     d_A[9] * d_U[u_idx + 2] + d_A[13] * d_U[u_idx + 3]) *
                    (d_A[3] * d_U[u_idx + 0] + d_A[7] * d_U[u_idx + 1] +
                     d_A[11] * d_U[u_idx + 2] + d_A[15] * d_U[u_idx + 3]) *
                    2.0;
  d_J[j_idx + 17] = (d_A[2] * d_U[u_idx + 0] + d_A[6] * d_U[u_idx + 1] +
                     d_A[10] * d_U[u_idx + 2] + d_A[14] * d_U[u_idx + 3]) *
                    (d_A[3] * d_U[u_idx + 0] + d_A[7] * d_U[u_idx + 1] +
                     d_A[11] * d_U[u_idx + 2] + d_A[15] * d_U[u_idx + 3]) *
                    2.0;
  d_J[j_idx + 18] = pow(d_A[3] * d_U[u_idx + 0] + d_A[7] * d_U[u_idx + 1] +
                            d_A[11] * d_U[u_idx + 2] + d_A[15] * d_U[u_idx + 3],
                        2.0) *
                    2.0;
  d_J[j_idx + 19] =
      -2.0 * (d_A[3] * d_U[u_idx + 0] + d_A[7] * d_U[u_idx + 1] +
              d_A[11] * d_U[u_idx + 2] + d_A[15] * d_U[u_idx + 3]);  // * y
}

__global__ void d_next_iteration(const float* d_J, const float* d_P,
                                 const float* d_raw_pixels,
                                 const bool* d_is_valid, float* d_GD,
                                 uint32_t* d_nums, const uint32_t P,
                                 const uint32_t W, const uint32_t N,
                                 const uint32_t column) {
  const uint32_t row = threadIdx.x;
  const uint32_t idx = row * W + column * (W / N);  // idx for pixel
  const uint32_t p_idx = row * (N * P + 1) + column * P;
  uint32_t u_idx = 0;

  assert(P < 10);
  uint32_t nums = 0;

  float J[10];
  for (uint32_t i = 0; i < P + 1; ++i) J[i] = 0.0f;

  for (uint32_t k = 0; k < W / N; ++k) {
    if (!d_is_valid[idx + k]) {
      u_idx += (P + 1) * (P + 2);
      continue;
    }

    nums++;

    for (uint32_t i = 0; i < P + 1; ++i) {
      for (uint32_t j = 0; j < P + 1; ++j) {
        J[i] += d_J[u_idx++] * d_P[p_idx + j];
      }
      J[i] += d_J[u_idx++] * d_raw_pixels[idx + k];
    }
  }

  for (uint32_t i = 0; i < P + 1; ++i) {
    d_GD[p_idx + i] += J[i];
    d_nums[p_idx + i] += nums;
  }
}

__global__ void d_update_P(const float* d_GD, const uint32_t* d_nums,
                           float* d_P, const float alpha) {
  const uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;

  d_P[idx] -= (alpha * d_GD[idx] / d_nums[idx]);
}

ImageProcessing::ImageProcessing(const uint32_t w, const uint32_t h,
                                 const uint32_t P, const uint32_t N)
    : m_size{w, h}, m_P(P), m_N(N) {
  d_A.resize((P + 1) * (P + 1));
  d_initialize_A<<<1, 1>>>(thrust::raw_pointer_cast(d_A.data()));
  hipDeviceSynchronize();
  d_U.resize((w / N) * (P + 1));
  d_initialize_U<<<w / N, P + 1>>>(thrust::raw_pointer_cast(d_U.data()), m_N);
  hipDeviceSynchronize();
  d_P.resize(h * (N * P + 1));
  d_GD.resize(h * (N * P + 1));
  d_nums.resize(h * (N * P + 1));
  d_pixels.resize(w * h);
  d_raw_pixels.resize(w * h);
  h_raw_pixels.resize(w * h);
  d_is_valid.resize(w * h);
  d_error.resize(w * h);
  d_J.resize((w / N) * (P + 1) * (P + 2));
  d_initialize_J<<<1, w / N>>>(thrust::raw_pointer_cast(d_J.data()),
                               thrust::raw_pointer_cast(d_U.data()),
                               thrust::raw_pointer_cast(d_A.data()), P);
  hipDeviceSynchronize();

  // thrust::host_vector<float> h_J;
  // h_J.resize(d_U.size());
  // thrust::copy(d_U.begin(), d_U.end(), h_J.begin());
  // for (uint32_t k = 0; k < 80; ++k) {
  //   std::cout << "i = " << k << "=============\n";
  //   for (uint32_t i = k * 4; i < (k + 1) * 4; ++i) {
  //     std::cout << std::fixed << std::setprecision(4) << h_J[i] << " ";
  //   }
  //   std::cout << '\n';
  // }

  // thrust::host_vector<float> h_J;
  // h_J.resize(d_J.size());
  // thrust::copy(d_J.begin(), d_J.end(), h_J.begin());
  // for (uint32_t k = 0; k < 80; ++k) {
  //   std::cout << "i = " << k << "=============\n";
  //   for (uint32_t i = k * 4; i < (k + 1) * 4; ++i) {
  //     for (uint32_t j = 0; j < 5; ++j)
  //       std::cout << std::fixed << std::setprecision(4) << h_J[i * 5 + j]
  //                 << " ";
  //     std::cout << '\n';
  //   }
  // }
}

void ImageProcessing::evaluate_pixels() {
  d_evaluate_pixels<<<m_size.y, m_size.x>>>(
      thrust::raw_pointer_cast(d_pixels.data()),
      thrust::raw_pointer_cast(d_U.data()),
      thrust::raw_pointer_cast(d_A.data()),
      thrust::raw_pointer_cast(d_P.data()), m_P, m_N);
  hipDeviceSynchronize();
}

void ImageProcessing::set_p(float p) {
  thrust::fill(thrust::device, d_P.begin(), d_P.end(), p);
}

void ImageProcessing::update_raw_pixels() {
  thrust::copy(h_raw_pixels.begin(), h_raw_pixels.end(), d_raw_pixels.begin());
}

void ImageProcessing::update_is_valid(float th) {
  d_update_is_valid<<<m_size.y, m_size.x>>>(
      thrust::raw_pointer_cast(d_is_valid.data()),
      thrust::raw_pointer_cast(d_raw_pixels.data()), th);
  hipDeviceSynchronize();
}

float ImageProcessing::evaluate_error() {
  d_evaluate_error<<<m_size.y, m_size.x>>>(
      thrust::raw_pointer_cast(d_is_valid.data()),
      thrust::raw_pointer_cast(d_raw_pixels.data()),
      thrust::raw_pointer_cast(d_pixels.data()),
      thrust::raw_pointer_cast(d_error.data()));
  hipDeviceSynchronize();

  return thrust::reduce(d_error.begin(), d_error.end(), 0.0f);
}

void ImageProcessing::fill_raw_pixels(float value) {
  thrust::fill(h_raw_pixels.begin(), h_raw_pixels.end(), value);
  update_raw_pixels();
}

void ImageProcessing::next_iteration(const float max_alpha) {
  d_clear<<<m_size.y, m_N * m_P + 1>>>(thrust::raw_pointer_cast(d_GD.data()),
                                       thrust::raw_pointer_cast(d_nums.data()));

  for (uint32_t i = 0; i < m_N; ++i) {
    d_next_iteration<<<1, m_size.y>>>(
        thrust::raw_pointer_cast(d_J.data()),
        thrust::raw_pointer_cast(d_P.data()),
        thrust::raw_pointer_cast(d_raw_pixels.data()),
        thrust::raw_pointer_cast(d_is_valid.data()),
        thrust::raw_pointer_cast(d_GD.data()),
        thrust::raw_pointer_cast(d_nums.data()), m_P, m_size.x, m_N, i);
    hipDeviceSynchronize();
  }

  float max_err = *thrust::max_element(d_error.begin(), d_error.end());
  d_update_P<<<m_size.y, m_N * m_P + 1>>>(
      thrust::raw_pointer_cast(d_GD.data()),
      thrust::raw_pointer_cast(d_nums.data()),
      thrust::raw_pointer_cast(d_P.data()),
      std::min(max_alpha, 0.01f * max_err));
  hipDeviceSynchronize();
}

void ImageProcessing::print() {
  for (uint32_t i = 0; i < d_pixels.size(); ++i)
    std::cout << '(' << i / m_size.x << ',' << i % m_size.x
              << ") : " << d_pixels[i] << '\n';
}

void ImageProcessing::savefile(const std::string name) {
  std::ofstream file;
  file.open(name);
  // for (uint32_t i = 0; i < d_nums.size(); ++i) file << d_nums[i] << ',';
  for (uint32_t i = 0; i < d_error.size(); ++i) file << d_error[i] << ',';
  file.close();
}
