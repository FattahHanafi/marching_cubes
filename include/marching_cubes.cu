#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <sys/types.h>
#include <thrust/detail/raw_pointer_cast.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/fill.h>
#include <thrust/reduce.h>

#include <algorithm>
#include <cstdint>
#include <cstdlib>
#include <cstring>
#include <iostream>
#include <iterator>
#include <ostream>
#include <vector>

#include "marching_cubes.hpp"

__global__ void d_update_vertices(bool* d_vertices, double* d_heights,
                                  const double size) {
  uint32_t vertex_idx = blockIdx.x * gridDim.y * blockDim.x +
                        blockIdx.y * blockDim.x + threadIdx.x;
  uint32_t height_idx = blockIdx.x * gridDim.y + blockIdx.y;
  d_vertices[vertex_idx] = (threadIdx.x * size) < d_heights[height_idx];
}

__global__ void d_update_cubes(uint8_t* d_cubes, bool* d_vertices) {
  uint32_t cube_idx = blockIdx.x * gridDim.y * blockDim.x +
                      blockIdx.y * blockDim.x + threadIdx.x;
  uint32_t vertex_idx[8];

  vertex_idx[0] = (blockIdx.x + 0) * (gridDim.y + 1) * (blockDim.x + 1) +
                  (blockIdx.y + 0) * (blockDim.x + 1) + (threadIdx.x + 0);
  vertex_idx[1] = (blockIdx.x + 1) * (gridDim.y + 1) * (blockDim.x + 1) +
                  (blockIdx.y + 0) * (blockDim.x + 1) + (threadIdx.x + 0);
  vertex_idx[2] = (blockIdx.x + 1) * (gridDim.y + 1) * (blockDim.x + 1) +
                  (blockIdx.y + 1) * (blockDim.x + 1) + (threadIdx.x + 0);
  vertex_idx[3] = (blockIdx.x + 0) * (gridDim.y + 1) * (blockDim.x + 1) +
                  (blockIdx.y + 1) * (blockDim.x + 1) + (threadIdx.x + 0);
  vertex_idx[4] = (blockIdx.x + 0) * (gridDim.y + 1) * (blockDim.x + 1) +
                  (blockIdx.y + 0) * (blockDim.x + 1) + (threadIdx.x + 1);
  vertex_idx[5] = (blockIdx.x + 1) * (gridDim.y + 1) * (blockDim.x + 1) +
                  (blockIdx.y + 0) * (blockDim.x + 1) + (threadIdx.x + 1);
  vertex_idx[6] = (blockIdx.x + 1) * (gridDim.y + 1) * (blockDim.x + 1) +
                  (blockIdx.y + 1) * (blockDim.x + 1) + (threadIdx.x + 1);
  vertex_idx[7] = (blockIdx.x + 0) * (gridDim.y + 1) * (blockDim.x + 1) +
                  (blockIdx.y + 1) * (blockDim.x + 1) + (threadIdx.x + 1);

  uint8_t cube = 0;

  for (uint8_t i = 0; i < 8; ++i) cube += (1 << i) * d_vertices[vertex_idx[i]];
  d_cubes[cube_idx] = cube;
}

__global__ void d_update_volumes(double* d_volumes, const uint8_t* d_cubes,
                                 const double* d_const_volumes) {
  uint32_t cube_idx = blockIdx.x * gridDim.y * blockDim.x +
                      blockIdx.y * blockDim.x + threadIdx.x;
  d_volumes[cube_idx] = d_volumes[d_cubes[cube_idx]];
}

MarchingCubes::MarchingCubes(const uint32_t x_count, const uint32_t y_count,
                             const uint32_t z_count, const double size)
    : m_count{x_count, y_count, z_count},
      m_cube_size{size, size, size},
      m_size{x_count * size, y_count * size, z_count * size} {
  d_vertices.resize((x_count + 1) * (y_count + 1) * (z_count + 1));
  d_cubes.resize(x_count * y_count * z_count);
  d_heights.resize((x_count + 1) * (y_count + 1));
  d_volumes.resize(x_count * y_count * z_count);
  d_const_volumes.resize(256);

  thrust::fill(d_heights.begin(), d_heights.end(), 0.0f);
  thrust::fill(d_vertices.begin(), d_vertices.end(), false);
  thrust::fill(d_cubes.begin(), d_cubes.end(), 0);
  thrust::fill(d_volumes.begin(), d_volumes.end(), 0.0);
  thrust::fill(d_volumes.begin(), d_volumes.end(), 0.5);  // UPDATE Values
}

MarchingCubes::~MarchingCubes() {}

void MarchingCubes::set_vertex(Vec3<uint32_t>* index, bool value) {
  uint32_t idx = vertex2idx(index);
  m_vertices->at(idx) = value;
}

uint32_t MarchingCubes::add() {
  return thrust::reduce(thrust::device, d_vertices.begin(), d_vertices.end(),
                        0);
}

bool MarchingCubes::get_vertex(const size_t i) const {
  return m_vertices->at(i);
}

size_t MarchingCubes::vertex2idx(const Vec3<uint32_t>* index) const {
  size_t idx = index->z;
  idx += index->y * (m_count.z + 1);
  idx += index->x * (m_count.z + 1) * (m_count.y + 1);
  return idx;
}

size_t MarchingCubes::cube2idx(const Vec3<uint32_t>* index) const {
  size_t idx = index->z;
  idx += index->y * m_count.z;
  idx += index->x * m_count.z * m_count.y;
  return idx;
}

void MarchingCubes::set_heights_gpu(double height) {
  thrust::fill(d_heights.begin(), d_heights.end(), height);
}

double MarchingCubes::update_volumes_gpu() {
  d_update_volumes<<<dim3(m_count.x, m_count.y, 1), m_count.z>>>(
      thrust::raw_pointer_cast(d_volumes.data()),
      thrust::raw_pointer_cast(d_cubes.data()),
      thrust::raw_pointer_cast(d_const_volumes.data()));
  hipDeviceSynchronize();
  return thrust::reduce(d_volumes.begin(), d_volumes.end(), 0.0);
}

void MarchingCubes::update_vertices_gpu() {
  d_update_vertices<<<dim3(m_count.x + 1, m_count.y + 1, 1), m_count.z + 1>>>(
      thrust::raw_pointer_cast(d_vertices.data()),
      thrust::raw_pointer_cast(d_heights.data()), m_cube_size.z);
  hipDeviceSynchronize();
}

void MarchingCubes::update_cubes_gpu() {
  d_update_cubes<<<dim3(m_count.x, m_count.y, 1), m_count.z>>>(
      thrust::raw_pointer_cast(d_cubes.data()),
      thrust::raw_pointer_cast(d_vertices.data()));
  hipDeviceSynchronize();
}

void MarchingCubes::print() {
  m_size.print();
  m_cube_size.print();
  m_size.print();
}
